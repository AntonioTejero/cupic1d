/****************************************************************************
 *                                                                          *
 *    CUPIC1D is a code that simulates the interaction between plasma and   *
 *    a langmuir probe in 1D using PIC techniques accelerated with the use  * 
 *    of GPU hardware (CUDA, extension of C/C++)                            *
 *                                                                          *
 ****************************************************************************/


/****************************** HEADERS ******************************/

#include "stdh.h"
#include "init.h"
#include "cc.h"
#include "mesh.h"
#include "particles.h"
#include "diagnostic.h"

/************************ FUNCTION PROTOTIPES *************************/




/*************************** MAIN FUNCTION ****************************/

int main (int argc, const char* argv[])
{
  /*--------------------------- function variables -----------------------*/
  
  // host variables definition
  double t;                             // time of simulation
  const double dt = init_dt();          // time step
  const int n_ini = init_n_ini();       // number of first iteration
  const int n_prev = init_n_prev();     // number of iterations before start analizing
  const int n_save = init_n_save();     // number of iterations between diagnostics
  const int n_fin = init_n_fin();       // number of last iteration
  int num_e, num_i;                     // number of particles (electrons and ions)
  char filename[50];                    // filename for saved data
  ifstream ifile;
  ofstream ofile;

  // device variables definition
  double *d_rho, *d_phi, *d_E;          // mesh properties
  particle *d_e, *d_i;                  // particles vectors
  hiprandStatePhilox4_32_10_t *state;    // philox state for __device__ random number generation 

  /*----------------------------- function body -------------------------*/

  // initialize device and simulation
  init_dev();
  init_sim(&d_rho, &d_phi, &d_E, &d_e, &num_e, &d_i, &num_i, &t, &state);

  cout << "t = " << t << endl;
  sprintf(filename, "../output/particles/electrons_t_%d", n_ini);
  particles_snapshot(d_e, filename, t);
  sprintf(filename, "../output/particles/ions_t_%d", n_ini);
  particles_snapshot(d_i, filename, t);
  t += dt;

  for (int i = n_ini+1; i <= n_fin; i++, t += dt) {
    // deposit charge into the mesh nodes
    charge_deposition(d_rho, d_e, d_i);
    cout << "Charge deposited" << endl;
    
    // solve poisson equation
    poisson_solver(1.0e-4, d_rho, d_phi);
    cout << "Poisson eq. solved" << endl;
    
    // derive electric fields from potential
    field_solver(d_phi, d_E);
    cout << "Fields soved" << endl;
    
    // move particles
    particle_mover(d_e, d_i, d_E);
    cout << "Particles moved" << endl;

    // contour condition
    cc(t, &d_e, &d_i, d_E, state);
    cout << "Contour conditions applied" << endl;

    // store data
    if (i>=n_prev && i%n_save==0) {
      sprintf(filename, "../output/particles/electrons_t_%d", i);
      particles_snapshot(d_e, filename, t);
      sprintf(filename, "../output/particles/ions_t_%d", i);
      particles_snapshot(d_i, filename, t);
      sprintf(filename, "../output/charge/charge_t_%d", i-1);
      mesh_snapshot(d_rho, filename);
      sprintf(filename, "../output/potential/potential_t_%d", i-1);
      mesh_snapshot(d_phi, filename);
      sprintf(filename, "../output/particles/bm_electrons_t_%d", i);
      save_bm(d_e_bm, filename);
      sprintf(filename, "../output/particles/bins_electrons_t_%d", i);
      save_bins(d_e_bm, d_e, filename);
    }
     
    // print simulation time
    cout << "t = " << t << endl;
  }

  ifile.open("../input/input_data");
  ofile.open("../input/input_data_new");
  if (ifile.is_open() && ofile.is_open()) {
    ifile.getline(filename, 50);
    ofile << filename << endl;
    ifile.getline(filename, 50);
    ofile << "n_ini = " << n_fin << ";" << endl;
    ifile.getline(filename, 50);
    while (!ifile.eof()) {
      ofile << filename << endl;
      ifile.getline(filename, 50);
    }
  }
  ifile.close();
  ofile.close();
  system("mv ../input/input_data_new ../input/input_data");
  
  cout << "Simulation finished!" << endl;
  return 0;
}
